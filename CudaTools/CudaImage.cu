#include "hip/hip_runtime.h"
#include "CudaImage.h"

#define LOG_CUDA


#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// Constructor
CudaImage::CudaImage()
{
	
}

CudaImage::~CudaImage()
{

}





/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Cuda Kernels

__global__ void Compute_Histogram_512_Cuda(uint32_t* hist, const uint16_t* data, uint16_t width, uint16_t height, uint8_t maxValueBitWidth)
{
	// NOTE: # of bins of histogram must match block size (number of threads in block), and in this case must be 512.
	//		 i.e. the number of threads per block must be the same as the number of bins.

	// maxValueBitWidth = the number of bits needed to represent the max value in the data array.  For example, if the data
	//					  array is built from a 10-bit A-to-D converter, then maxValueBitWidth = 10 since no value will be greather 
	//					  than 2^10.  The minimum value for maxValueBitWidth is driven by the number of bins.  For 256 bins (2^8), 
	//					  the min value is 8.  If bins were 1024 (i.e. 2^10), then the min value for maxValueBitWidth would be 10.

	if (maxValueBitWidth < 8) maxValueBitWidth = 8; // make sure we aren't below the min as described above

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int nThread = threadIdx.y * blockDim.x + threadIdx.x; // index of thread within block
	int nPixel = y * width + x; // index of pixel within image

	if (x >= width) return;
	if (y >= height) return;

	// if image pixel value == 0, don't add it to the histogram.  Pixel that are 0 are pixels that are outside of the mask 
	// and thus should not be part of the histogram
	if (data[nPixel] == 0) return;

	//Create shared buffer size of threads per block and clear it 
	//Size of array equals numBins 
	__shared__ uint32_t tmpHist[512];
	tmpHist[nThread] = 0;
	__syncthreads();


	//based on the value of this pixel, find the correct bin of the local histogram to increment, and then increment it
	uint8_t shift = maxValueBitWidth - 9;
	int binNumber = data[nPixel] >> shift;

	if (binNumber>511)
	{
		binNumber = 511;
	}

	//float f1 = ((float)(data[nPixel]))/1023.0 * 255;
	//uint8_t binNumber = (uint8_t)f1;


	atomicAdd(&(tmpHist[binNumber]), 1);
	__syncthreads();  // wait for all threads in this block to finish so that the local histogram is finished

					  // Update global memory (global histogram)	
	atomicAdd(&(hist[nThread]), tmpHist[nThread]);

}

__global__ void compute_histogram_256_Cuda(uint32_t* hist, const uint16_t* data, uint16_t width, uint16_t height, uint8_t maxValueBitWidth)
{
	// NOTE: # of bins of histogram must match block size (number of threads in block), and in this case must be 256.
	//		 i.e. the number of threads per block must be the same as the number of bins.

	// maxValueBitWidth = the number of bits needed to represent the max value in the data array.  For example, if the data
	//					  array is built from a 10-bit A-to-D converter, then maxValueBitWidth = 10 since no value will be greather 
	//					  than 2^10.  The minimum value for maxValueBitWidth is driven by the number of bins.  For 256 bins (2^8), 
	//					  the min value is 8.  If bins were 1024 (i.e. 2^10), then the min value for maxValueBitWidth would be 10.

	if (maxValueBitWidth < 8) maxValueBitWidth = 8; // make sure we aren't below the min as described above

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int nThread = threadIdx.y * blockDim.x + threadIdx.x; // index of thread within block
	int nPixel = y * width + x; // index of pixel within image

	if (x >= width) return;
	if (y >= height) return;

	//Create shared buffer size of threads per block and clear it 
	//Size of array equals numBins 
	__shared__ uint32_t tmpHist[256];
	tmpHist[nThread] = 0;
	__syncthreads();


	//based on the value of this pixel, find the correct bin of the local histogram to increment, and then increment it
	uint8_t shift = maxValueBitWidth - 8;
	int binNumber = data[nPixel] >> shift;

	if (binNumber>255)
	{
		binNumber = 255;
	}

	//float f1 = ((float)(data[nPixel]))/1023.0 * 255;
	//uint8_t binNumber = (uint8_t)f1;


	atomicAdd(&(tmpHist[binNumber]), 1);
	__syncthreads();  // wait for all threads in this block to finish so that the local histogram is finished

					  // Update global memory (global histogram)	
	atomicAdd(&(hist[nThread]), tmpHist[nThread]);

}

__global__ void MaskImage_Cuda(uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	// this function zeroes out all pixels in image that are not in the mask

	// image - a greyscale image with each pixel being a uint16_t
	// mask - a image where pixels with value>0 will be passed through, and pixels with value==0 will be masked out (set to zero).
	//		  The mask is created where pixels with a value of 1, belong in mask aperture 1.  Pixels with value of 2, belong in 
	//		  mask aperture 2...and so on.  
	// width,height - dimensions of image in pixels

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in array
	uint32_t n = (y * width) + x;

	// apply mask to image
	if (mask[n] == 0)
	{
		// this pixel is not within a mask aperture, so zero it out
		image[n] = 0;
	}
}

__global__ void FlattenImage_Cuda(uint16_t* image, float* Gc, float* Dc, uint16_t width, uint16_t height)
{
	// this function flattens the image using

	// image - a greyscale image with each pixel being a uint16_t	
	// width,height - dimensions of image in pixels
	// Equation:
	//              flattenedImage[n] = (inputImage[n] - Dc[n]) * Gc[n]
	//	
	//
	//  C = corrected image  (Cij = the pixel at column i and row j)	
	//  D = dark image (this is an image taken with no lighting.  it bascially gives the dark current noise)
	//  G = gain
	//  Dc = D corrected to binning size
	//  Gc = G corrected to binning size


	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in array
	uint32_t n = (y * width) + x;

	// adjust pixel to flatten image
	float fval = ((float)image[n] - Dc[n]) * Gc[n];
	if (fval < 0.0f) fval = 0.0f;
	if (fval > 65535.0f) fval = 65535.0f;

	image[n] = (uint16_t)fval;
}


__global__ void ConvertGrayscaleToColor_Cuda(uint8_t* color, uint16_t* gray, uint8_t* redMap, uint8_t* greenMap, uint8_t* blueMap,
	uint16_t width, uint16_t height, uint16_t maxGrayValue, uint16_t scaleLower, uint16_t scaleUpper)
{
	// this function converts a grayscale image to a color image using the provided color map

	// color - destination color image (format is ARGB)
	// gray -  source grayscale image
	// redMap, greenMap, blueMap - arrays (maps) that provide color components for each possible grayscale value. For example,
	//							   if a pixel in the gray image has a value = 100, then the corresponding pixel in the color image
	//							   would have its RGB component values set to redMap[100], greenMap[100], and blueMap[100], respectively.
	// width, height - image dimensions
	// maxGrayValue - the maximum possible grayscale value, i.e. length of color map (length of redMap, greenMap, and blueMap)

	// scaleLower, scaleUpper - these values are used to scale the grayscale value of a pixel before it is converted to color.
	//
	//                         scaleUpper
	//						   ________________
	//	maxGrayValue|         /
	//				|        /
	//				|       /
	//				|      /
	//			0	|_____/____________________ 
	//                   scaleLower
	//
	//  Here's the math:
	//		if (pixelValue < scaleLower) set pixelValue = 0
	//      else if (pixelValue < scaleUpper) set pixelValue = maxGrayValue
	//      else 

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in gray array
	uint32_t nG = (y * width) + x;

	// calculate pixel position in color array
	uint32_t nC = (y * width * 4) + (x * 4);

	// make sure grayscale value is not outside of color maps
	if (gray[nG] > maxGrayValue) gray[nG] = maxGrayValue;

	// scale the value
	uint16_t val = gray[nG];
	if (val < scaleLower) val = 0;
	else if (val >= scaleUpper) val = maxGrayValue;
	else
	{
		float fval = (float)maxGrayValue / (float)(scaleUpper - scaleLower) * (float)(val - scaleLower);
		val = (uint16_t)fval;
	}

	// set pixel component values for color image
	color[nC + 0] = blueMap[val];	// blue
	color[nC + 1] = greenMap[val];	// green
	color[nC + 2] = redMap[val];	// red
	color[nC + 3] = 255;			// alpha

}

__global__ void CopyCudaArrayToD3D9Memory_Cuda(uint8_t *dest, uint8_t *source, uint16_t pitch, uint16_t width, uint16_t height)
{
	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calc position of pixel in cuda array (remember that pitch may not equal width)
	//uint32_t nD = ((height - 1 - y)*pitch) + (x * 4);
	uint32_t nD = (y*pitch) + (x * 4);
	uint32_t nS = (y*width * 4) + (x * 4);

	// copy data
	dest[nD] = source[nS];
	dest[nD + 1] = source[nS + 1];
	dest[nD + 2] = source[nS + 2];
	dest[nD + 3] = source[nS + 3];
}

__global__ void BuildHistogramImage_Cuda(uint8_t* histImage, uint32_t* hist, uint16_t numBins, uint16_t width, uint16_t height, uint32_t maxBinCount)
{
	// this function builds the image for a histogram given by the variable hist.  
	//
	// histImage - the output histogram image.  This is a color image (ARGB, 8 bits per component)
	// hist - is an array which contains the data for the histogram
	// numBins - is the number of bins in the histogram
	// width, height - dimensions of the histImage in pixels
	// maxBinCount - the maximum value that can appear in each bin of the histogram

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate the array index into the histogram image
	uint32_t n = (y * width * 4) + (x * 4);  // ARGB image

											 // calculate the width of each bin in pixels
	uint16_t binWidth = width / numBins;

	// calculate the bin that this pixel belongs in
	uint16_t binNumber = x / binWidth;
	if (binNumber>numBins) binNumber = numBins;

	// calculate height of the bar for his bin
	uint32_t value = hist[binNumber];  // get the height of the bar for this bin
	uint32_t barHeight = (uint32_t)((float)value * (float)height / (float)maxBinCount);  // calculate the bar height in pixels
	if (barHeight > height) barHeight = height; // make sure the bar height in pixels is not greater than the histogram image height

												// determine if this pixel is in the bar or above it (i.e. determine color of pixel)
	if (y < (height - barHeight)) // pixel is above bar (thus pixel is background color...likely white)
	{
		histImage[n + 0] = 220;	// blue
		histImage[n + 1] = 220;	// green
		histImage[n + 2] = 220;	// red
		histImage[n + 3] = 255;	// alpha
	}
	else  // pixel is part of bar, so make it the color of the bar (likely black)
	{
		histImage[n + 0] = 0;	// blue
		histImage[n + 1] = 0;	// green
		histImage[n + 2] = 0;	// red
		histImage[n + 3] = 255;	// alpha
	}
}

__global__ void CalcApertureSums_Cuda(uint32_t* sumArray, uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	// This function calculate the sum of pixels for each aperture of a mask.  It expects that the mask is formated as follows:
	//		mask pixels with a value of 0 belong to no apertures, thus they will not be part of any sum
	//      mask pixels with a value of 1 belong in aperture 1, which is added to the value in sumArray[0]
	//      mask pixels with a value of 2 belong in aperture 2, which is added to the value in sumArray[1]
	//		and so on...

	// sumArray - output array of the sum of pixel values for each aperature.  For example, for a mask with 24x16 (384) apertures, there
	//			  will be 384 values in sumArray
	// image - input grayscale image from which sums are calculated
	// mask  - input mask that is formatted as described in the description above for this function
	// width, height - dimensions of the image and mask in pixels

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in image and mask
	uint32_t n = (y * width) + x;

	// get aperture number from mask
	if (mask[n] > 0) // is this pixel inside of any of the apertures of the mask?
	{ // yes
		atomicAdd(&sumArray[mask[n] - 1], image[n]);
	}

	__syncthreads();  // wait for all threads in this block to finish so that the local histogram is finished

}

__global__ void FlatField_Cuda(uint16_t* image, uint16_t* dark, uint16_t* gain, uint16_t width, uint16_t height)
{
	// this function flat field corrects the given grayscale image. It uses the following function:
	//
	//		C[i,j] = ((R[i,j] - D[i,j]) * m) / (F[i,j] - D[i,j]) = (R[i,j] - D[i,j]) * G[i,j]
	//
	//			where G[i,j] = m / (F[i,j] - D[i,j])
	//
	//				  m = average of F-D
	//
	//		i,j = row,column of pixel in image
	//		C = corrected image
	//		R = raw image
	//		F = flat field reference image (evenly illuminated image, meant to show unevenness of illumination)
	//		D = dark field reference image (image taken with no illumination, meant to show distribution of background)
	//		G = gain

	//	parameters passed into function:
	//	image - grayscale image to be corrected.  This is both the input and output image (the input image is over written)
	//  dark  - this is the dark field image (must be same dimensions as image), probably stored in database
	//  gain  - this is the gain array (must be same dimensions as image), that is calculated elsewhere
	//  width, height - dimensions of image (and dark) in pixels

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in image and dark arrays
	uint32_t n = (y * width) + x;

	image[n] = (image[n] - dark[n]) * gain[n];
}

__global__ void CopyRoiToFullImage_Cuda(uint16_t* full, uint16_t* roi, uint16_t fullW, uint16_t fullH,
	uint16_t  roiX, uint16_t roiY, uint16_t roiW, uint16_t roiH)
{
	// This function is used to copy a ROI image from the camera into a memory space that holds a full frame.
	// It is used when the camera is set up to capture only a part of the CCD (an Region of Interest - ROI), and 
	// since all of the algorithms, kernels, display routines, etc. are set up to handle full frames, this
	// function simply copies the ROI into a full frame.  Pixels outside the ROI are set to zero.

	// calc x,y position of pixel to operate on in the full frame
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside full frame image
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside full frame image

														// make sure we don't try to operate outside the full image
	if (x >= fullW) return;
	if (y >= fullH) return;

	// calculate pixel position in arrays
	uint32_t fullN = (y * fullW) + x;  // index into full frame

									   // calculate x,y position in ROI
	int32_t xr = x - roiX;
	int32_t yr = y - roiY;

	// are we inside ROI?

	if (x >= roiX && x < (roiX + roiW) && y >= roiY && y < (roiY + roiH))
	{
		uint32_t roiN = (yr * roiW) + xr; // index into roi frame

										  // inside ROI
		full[fullN] = roi[roiN];
	}
	else
	{
		// outside ROI
		full[fullN] = 0;
	}
}




template <unsigned int blockSize> __device__ void warpReduce(volatile uint64_t *sdata, unsigned int tid)
{
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8)  sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4)  sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2)  sdata[tid] += sdata[tid + 1];
}


template <unsigned int blockSize> __global__ void reduce6(uint16_t *g_idata, uint64_t *g_odata, unsigned int n)
{
	extern __shared__ uint64_t sdata[blockSize];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;

	while (i < n)
	{
		sdata[tid] += g_idata[i] + g_idata[i + blockSize];  
		i += gridSize;
	}
	__syncthreads();

	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			sdata[tid] += sdata[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			sdata[tid] += sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128)
	{
		if (tid < 64)
		{
			sdata[tid] += sdata[tid + 64];
		}
		__syncthreads();
	}
	if (tid < 32) warpReduce<blockSize>(sdata, tid);
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
	__device__ inline operator T *()
	{
		extern __shared__ int __smem[];
		return (T *)__smem;
	}

	__device__ inline operator const T *() const
	{
		extern __shared__ int __smem[];
		return (T *)__smem;
	}
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
	__device__ inline operator double *()
	{
		extern __shared__ double __smem_d[];
		return (double *)__smem_d;
	}

	__device__ inline operator const double *() const
	{
		extern __shared__ double __smem_d[];
		return (double *)__smem_d;
	}
};


/*
This version adds multiple elements per thread sequentially.  This reduces the overall
cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
(Brent's Theorem optimization)

Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T1, class T2, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce6(T1 *g_idata, T2 *g_odata, unsigned int n)
{
	// Handle to thread block group
	cg::thread_block cta = cg::this_thread_block();
	T2 *sdata = SharedMemory<T2>();

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize * 2 + threadIdx.x;
	unsigned int gridSize = blockSize * 2 * gridDim.x;

	T2 mySum = 0;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n)
	{
		mySum += g_idata[i];

		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2 || i + blockSize < n)
			mySum += g_idata[i + blockSize];

		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = mySum;
	cg::sync(cta);


	// do reduction in shared mem
	if ((blockSize >= 512) && (tid < 256))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 256];
	}

	cg::sync(cta);

	if ((blockSize >= 256) && (tid < 128))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 128];
	}

	cg::sync(cta);

	if ((blockSize >= 128) && (tid <  64))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 64];
	}

	cg::sync(cta);

#if (__CUDA_ARCH__ >= 300 )
	if (tid < 32)
	{
		cg::coalesced_group active = cg::coalesced_threads();

		// Fetch final intermediate sum from 2nd warp
		if (blockSize >= 64) mySum += sdata[tid + 32];
		// Reduce final warp using shuffle
		for (int offset = warpSize / 2; offset > 0; offset /= 2)
		{
			mySum += active.shfl_down(mySum, offset);
		}
	}
#else
	// fully unroll reduction within a single warp
	if ((blockSize >= 64) && (tid < 32))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 32];
	}

	cg::sync(cta);

	if ((blockSize >= 32) && (tid < 16))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 16];
	}

	cg::sync(cta);

	if ((blockSize >= 16) && (tid <  8))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 8];
	}

	cg::sync(cta);

	if ((blockSize >= 8) && (tid <  4))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 4];
	}

	cg::sync(cta);

	if ((blockSize >= 4) && (tid <  2))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 2];
	}

	cg::sync(cta);

	if ((blockSize >= 2) && (tid <  1))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 1];
	}

	cg::sync(cta);
#endif

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = mySum;
}



bool isPow2(unsigned int x)
{
	return ((x&(x - 1)) == 0);
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T1, class T2>  // T1 is input type, T2 is output type
void
reduce(int size, int threads, int blocks, T1 *d_idata, T2 *d_odata)
{
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	// when there is only one warp per block, we need to allocate two warps
	// worth of shared memory so that we don't index shared memory out of bounds
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(T2) : threads * sizeof(T2);


		if (isPow2(size))
		{
			switch (threads)
			{
			case 512:
				reduce6<T1, T2, 512, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 256:
				reduce6<T1, T2, 256, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 128:
				reduce6<T1, T2, 128, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 64:
				reduce6<T1, T2, 64, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 32:
				reduce6<T1, T2, 32, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 16:
				reduce6<T1, T2, 16, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case  8:
				reduce6<T1, T2, 8, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case  4:
				reduce6<T1, T2, 4, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case  2:
				reduce6<T1, T2, 2, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case  1:
				reduce6<T1, T2, 1, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;
			}
		}
		else
		{
			switch (threads)
			{
			case 512:
				reduce6<T1, T2, 512, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 256:
				reduce6<T1, T2, 256, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 128:
				reduce6<T1, T2, 128, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 64:
				reduce6<T1, T2, 64, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 32:
				reduce6<T1, T2, 32, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case 16:
				reduce6<T1, T2, 16, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case  8:
				reduce6<T1, T2, 8, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case  4:
				reduce6<T1, T2, 4, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case  2:
				reduce6<T1, T2, 2, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;

			case  1:
				reduce6<T1, T2, 1, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
				break;
			}
		}

}



uint64_t CudaImage::SumImage(uint16_t* grayImage, uint16_t width, uint16_t height)
{
	unsigned int elementCount = ((unsigned int)width)*((unsigned int)height);
	unsigned int blockSize = 256;  // number of threads
	unsigned int numBlocks = (elementCount + blockSize - 1) / blockSize;

	// copy gray (input) image to gpu
	uint16_t* d_grayImage;
	hipError_t res = hipMalloc(&d_grayImage, elementCount * sizeof(uint16_t));
	res = hipMemcpy(d_grayImage, grayImage, elementCount * sizeof(uint16_t), hipMemcpyHostToDevice);

	// allocate for output data (of type uint64_t), and initialize to zero
	uint64_t* d_output;
	res = hipMalloc(&d_output, numBlocks * sizeof(uint64_t));
	hipMemset(d_output, 0, numBlocks * sizeof(uint64_t));

	reduce<uint16_t, uint64_t> (elementCount, blockSize, numBlocks, d_grayImage, d_output);

	uint64_t *p_vals = (uint64_t*)malloc(numBlocks*sizeof(uint64_t));

	res = hipMemcpy(p_vals, d_output, numBlocks * sizeof(uint64_t), hipMemcpyDeviceToHost);

	uint64_t sum = 0;

	for (int i = 0; i < numBlocks; i++)
		sum += p_vals[i];


	hipFree(d_grayImage);
	hipFree(d_output);
	free(p_vals);

	return sum;
}


uint64_t CudaImage::SumLoadedGrayImage()
{
	if (mp_d_grayImage == 0) return 0;

	unsigned int elementCount = ((unsigned int)m_imageW)*((unsigned int)m_imageH);
	unsigned int blockSize = 512;  // number of threads
	unsigned int numBlocks = (elementCount + blockSize - 1) / blockSize;

	// allocate for output data (of type uint64_t), and initialize to zero
	uint64_t* d_output;
	hipError_t res = hipMalloc(&d_output, numBlocks * sizeof(uint64_t));
	hipMemset(d_output, 0, numBlocks * sizeof(uint64_t));

	reduce<uint16_t, uint64_t>(elementCount, blockSize, numBlocks, mp_d_grayImage, d_output);

	uint64_t *p_vals = (uint64_t*)malloc(numBlocks * sizeof(uint64_t));

	res = hipMemcpy(p_vals, d_output, numBlocks * sizeof(uint64_t), hipMemcpyDeviceToHost);

	uint64_t sum = 0;

	for (int i = 0; i < numBlocks; i++)
		sum += p_vals[i];

	hipFree(d_output);
	free(p_vals);

	return sum;
}


void CudaImage::Test()
{
	uint16_t w = 1024;
	uint16_t h = 1024;

	uint16_t* p_data = (uint16_t*)malloc(w*h * sizeof(uint16_t));

	for (int r = 0; r < h; r++)
	{
		for (int c = 0; c < w; c++)
		{
			int index = r*w + c;

			p_data[index] = 1;
		}
	}


	GpuTimer t1;

	t1.Start();
	uint64_t sum = SumImage(p_data, w, h);
	t1.Stop();
	float t = t1.ElapsedMillis();

	free(p_data);

	float avg = (float)sum / (float)(w*h);

	sum += 0;


}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel Calling Functions

void CudaImage::ConvertGrayscaleToColor(uint8_t* color, uint16_t* gray, uint8_t* redMap, uint8_t* greenMap, uint8_t* blueMap,
	uint16_t width, uint16_t height, uint16_t maxGrayValue, uint16_t scaleLower, uint16_t scaleUpper)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	ConvertGrayscaleToColor_Cuda << <grid, block >> >(color, gray, redMap, greenMap, blueMap, width, height, maxGrayValue, scaleLower, scaleUpper);

}

void CudaImage::CopyRoiToFullImage(uint16_t* full, uint16_t* roi, uint16_t fullW, uint16_t fullH,
	uint16_t  roiX, uint16_t roiY, uint16_t roiW, uint16_t roiH)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = fullW / block.x;
	grid.y = fullH / block.y;
	grid.z = 1;
	CopyRoiToFullImage_Cuda << <grid, block >> >(full, roi, fullW, fullH, roiX, roiY, roiW, roiH);
}

void CudaImage::MaskImage(uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;
	MaskImage_Cuda << <grid, block >> >(image, mask, width, height);
}

void CudaImage::FlattenImage(uint16_t* image, float* Gc, float* Dc, uint16_t width, uint16_t height)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;
	FlattenImage_Cuda << <grid, block >> >(image, Gc, Dc, width, height);
}

void CudaImage::CopyCudaArrayToD3D9Memory(uint8_t* pDest, uint8_t* pSource, uint16_t pitch, uint16_t width, uint16_t height)
{
	hipError_t res = hipDeviceSynchronize();

	dim3 threadsPerBlock(32, 32);  // 32x16 = 512 threads per block	
	dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
	CopyCudaArrayToD3D9Memory_Cuda << <numBlocks, threadsPerBlock >> >(pDest, pSource, pitch, width, height);
}

void CudaImage::ComputeHistogram_512(uint32_t* hist, const uint16_t* data, uint16_t width, uint16_t height, uint8_t maxValueBitWidth)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1; // block size must be 512 = 32 * 16
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	Compute_Histogram_512_Cuda << <grid, block >> >(hist, data, width, height, maxValueBitWidth);
}

void CudaImage::BuildHistogramImage_512(uint8_t* histImage, uint32_t* hist, uint16_t numBins, uint16_t width, uint16_t height, uint32_t maxBinCount)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	BuildHistogramImage_Cuda << <grid, block >> >(histImage, hist, numBins, width, height, maxBinCount);
}

void CudaImage::CalcApertureSums(uint32_t* sumArray, uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	CalcApertureSums_Cuda << <grid, block >> >(sumArray, image, mask, width, height);
}





uint16_t* CudaImage::SetFullGrayscaleImage(uint16_t* grayImage, uint16_t imageWidth, uint16_t imageHeight)
{
	if (imageWidth != m_imageW || imageHeight != m_imageH)
	{
		if (mp_d_grayImage != 0) hipFree(mp_d_grayImage);
		if (mp_d_colorImage != 0) hipFree(mp_d_colorImage);

		m_imageW = imageWidth;
		m_imageH = imageHeight;
		hipError_t res = hipMalloc(&mp_d_grayImage, m_imageW*m_imageH * sizeof(uint16_t));
		res = hipMalloc(&mp_d_colorImage, m_imageW*m_imageH * 4);
	}

	hipError_t err = hipMemcpy(mp_d_grayImage, grayImage, m_imageW*m_imageH * sizeof(uint16_t), hipMemcpyHostToDevice);

	return mp_d_grayImage;
}



uint16_t* CudaImage::SetRoiGrayscaleImage(uint16_t* roiImage, uint16_t imageWidth, uint16_t imageHeight, uint16_t roiWidth, 
							   uint16_t roiHeight, uint16_t roiX, uint16_t roiY)
{
	if (imageWidth != m_imageW || imageHeight != m_imageH)
	{
		if (mp_d_grayImage != 0) hipFree(mp_d_grayImage);
		if (mp_d_colorImage != 0) hipFree(mp_d_colorImage);

		m_imageW = imageWidth;
		m_imageH = imageHeight;
		hipError_t res = hipMalloc(&mp_d_grayImage, m_imageW*m_imageH * sizeof(uint16_t));
		res = hipMalloc(&mp_d_colorImage, m_imageW*m_imageH * 4);
	}

	if (roiWidth != m_roiW || roiHeight != m_roiH || roiX != m_roiX || roiY != m_roiY)
	{
		if (mp_d_roiImage != 0) hipFree(mp_d_roiImage);

		m_roiW = roiWidth;
		m_roiH = roiHeight;
		m_roiX = roiX;
		m_roiY = roiY;
		hipMalloc(&mp_d_roiImage, m_roiW*m_roiH * sizeof(uint16_t));
	}

	hipError_t errNo = hipMemcpy(mp_d_roiImage, roiImage, m_roiW*m_roiH * sizeof(uint16_t), hipMemcpyHostToDevice);

	CopyRoiToFullImage(mp_d_grayImage, mp_d_roiImage, m_imageW, m_imageH, m_roiX, m_roiY, m_roiW, m_roiH);

	return mp_d_grayImage;
}



uint16_t* CudaImage::SetMaskImage(uint16_t* maskImage, uint16_t maskWidth, uint16_t maskHeight, uint16_t maskRows, uint16_t maskCols)
{
	if (m_maskW != maskWidth || m_maskH != maskHeight)
	{
		if (mp_d_maskImage != 0) hipFree(mp_d_maskImage);

		m_maskW = maskWidth;
		m_maskH = maskHeight;
		m_maskRows = maskRows;
		m_maskCols = maskCols;
		hipMalloc(&mp_d_maskImage, m_maskW*m_maskH * sizeof(uint16_t));
	}


	// copy mask image to GPU
	hipMemcpy(mp_d_maskImage, maskImage, m_maskW*m_maskH * sizeof(uint16_t), hipMemcpyHostToDevice);

	m_maskSet = true;

	return mp_d_maskImage;
}



void CudaImage::SetColorMap(uint8_t* redMap, uint8_t* greenMap, uint8_t* blueMap, uint16_t maxPixelValue)
{
	hipError_t res;
	if (mp_d_redMap != 0) hipFree(mp_d_redMap);
	if (mp_d_greenMap != 0) hipFree(mp_d_greenMap);
	if (mp_d_blueMap != 0) hipFree(mp_d_blueMap);

	res = hipMalloc(&mp_d_redMap, maxPixelValue + 1);
	res = hipMalloc(&mp_d_greenMap, maxPixelValue + 1);
	res = hipMalloc(&mp_d_blueMap, maxPixelValue + 1);

	res = hipMemcpy(mp_d_redMap, redMap, maxPixelValue + 1, hipMemcpyHostToDevice);
	res = hipMemcpy(mp_d_greenMap, greenMap, maxPixelValue + 1, hipMemcpyHostToDevice);
	res = hipMemcpy(mp_d_blueMap, blueMap, maxPixelValue + 1, hipMemcpyHostToDevice);

	m_maxPixelValue = maxPixelValue;

	m_colorMapSet = true;
}


uint8_t* CudaImage::ConvertGrayscaleToColor(uint16_t scaleLower, uint16_t scaleUpper)
{
	if (m_colorMapSet && mp_d_grayImage != 0)
	{
		if (mp_d_colorImage == 0) hipMalloc(&mp_d_colorImage, m_imageW*m_imageH * 4);

		ConvertGrayscaleToColor(mp_d_colorImage, mp_d_grayImage, mp_d_redMap, mp_d_greenMap, mp_d_blueMap, m_imageW, m_imageH, m_maxPixelValue, scaleLower, scaleUpper);
	}

	return mp_d_colorImage;
}


void CudaImage::ApplyMaskToImage()
{
	if (m_maskSet)
	{
		MaskImage(mp_d_grayImage, mp_d_maskImage, m_imageW, m_imageH);
	}
}


uint8_t* CudaImage::PipelineFullImage(uint16_t* grayImage, uint16_t imageWidth, uint16_t imageHeight, bool applyMask)
{
	SetFullGrayscaleImage(grayImage, imageWidth, imageHeight);
	if (applyMask) ApplyMaskToImage();
	ConvertGrayscaleToColor(0, m_maxPixelValue);

	return mp_d_colorImage;
}


uint8_t* CudaImage::PipelineRoiImage(uint16_t* roiImage, uint16_t imageWidth, uint16_t imageHeight, 
	uint16_t roiWidth, uint16_t roiHeight, uint16_t roiX, uint16_t roiY, bool applyMask)
{
	SetRoiGrayscaleImage(roiImage, imageWidth, imageHeight, roiWidth, roiHeight, roiX, roiY);
	if (applyMask) ApplyMaskToImage();
	ConvertGrayscaleToColor(0, m_maxPixelValue);

	return mp_d_colorImage;
}


void CudaImage::Init()
{
	mp_d_grayImage = 0;
	mp_d_colorImage = 0;
	mp_d_maskImage = 0;
	mp_d_roiImage = 0;
	mp_d_redMap = 0;
	mp_d_greenMap = 0;
	mp_d_blueMap = 0;
	m_colorMapSet = false;
	m_maskSet = false;
	m_imageW = 0;
	m_imageH = 0;
	m_roiW = 0;
	m_roiH = 0;
	m_roiX = 0;
	m_roiY = 0;
	m_maskW = 0;
	m_maskH = 0;
	m_maskRows = 0;
	m_maskCols = 0;
	m_maxPixelValue = 65535;
	mp_d_histogram = 0;
	mp_d_colorHistogramImage = 0;

	mp_d_FFC_Fluor_Gc = 0;
	mp_d_FFC_Fluor_Dc = 0;
	mp_d_FFC_Lumi_Gc = 0;
	mp_d_FFC_Lumi_Dc = 0;
	m_h_FFC_numElements = 0;


	// not sure why I have to do this, bu
	hipMalloc(&mp_d_grayImage, 10);
	hipMalloc(&mp_d_colorImage, 10);
}


void CudaImage::Shutdown()
{
	if (this != 0)
	{
		if (mp_d_grayImage != 0) {
			hipError_t err = hipFree(mp_d_grayImage);
			mp_d_grayImage = 0;
		}
		if (mp_d_colorImage != 0) {
			hipFree(mp_d_colorImage);
			mp_d_colorImage = 0;
		}
		if (mp_d_maskImage != 0) {
			hipFree(mp_d_maskImage);
			mp_d_maskImage = 0;
		}
		if (mp_d_roiImage != 0) {
			hipFree(mp_d_roiImage);
			mp_d_roiImage = 0;
		}
		if (mp_d_redMap != 0) {
			hipFree(mp_d_redMap);
			mp_d_redMap = 0;
		}
		if (mp_d_greenMap != 0) {
			hipFree(mp_d_greenMap);
			mp_d_greenMap = 0;
		}
		if (mp_d_blueMap != 0) {
			hipFree(mp_d_blueMap);
			mp_d_blueMap = 0;
		}
		if (mp_d_histogram != 0) {
			hipFree(mp_d_histogram);
			mp_d_histogram = 0;
		}
		if (mp_d_colorHistogramImage != 0) {
			hipFree(mp_d_colorHistogramImage);
			mp_d_colorHistogramImage = 0;
		}
		if (mp_d_maskApertureSums != 0) {
			hipFree(mp_d_maskApertureSums);
			mp_d_maskApertureSums = 0;
		}
		if (mp_d_FFC_Fluor_Gc != 0) {
			hipFree(mp_d_FFC_Fluor_Gc);
			mp_d_FFC_Fluor_Gc = 0;
		}
		if (mp_d_FFC_Fluor_Dc != 0) {
			hipFree(mp_d_FFC_Fluor_Dc);
			mp_d_FFC_Fluor_Dc = 0;
		}
		if (mp_d_FFC_Lumi_Gc != 0) {
			hipFree(mp_d_FFC_Lumi_Gc);
			mp_d_FFC_Lumi_Gc = 0;
		}
		if (mp_d_FFC_Lumi_Dc != 0) {
			hipFree(mp_d_FFC_Lumi_Dc);
			mp_d_FFC_Lumi_Dc = 0;
		}
	}
}


void CudaImage::GetHistogram_512Buckets(uint32_t* destHist, uint8_t maxValueBitWidth)
{
	if (mp_d_histogram == 0)
	{
		hipMalloc(&mp_d_histogram, 512 * sizeof(uint32_t));
	}

	hipMemset(mp_d_histogram, 0, 512 * sizeof(uint32_t));

	ComputeHistogram_512(mp_d_histogram, mp_d_grayImage, m_imageW, m_imageH, maxValueBitWidth);

	//hipMemset(mp_d_histogram, 0, sizeof(uint32_t));  // zero the first bin, since that is the pixels that were masked out

	hipMemcpy(destHist, mp_d_histogram, 512 * sizeof(uint32_t), hipMemcpyDeviceToHost);

	m_max_histogramBinValue = 0;

	for (int i = 1; i < 512; i++)
	{
		if (destHist[i] > m_max_histogramBinValue) m_max_histogramBinValue = destHist[i];
	}

}


void CudaImage::GetHistogramImage_512Buckets(uint8_t* histImage, uint16_t width, uint16_t height, uint32_t maxBinCount)
{
	// NOTE:  GetHistogram_512Buckets MUST BE CALLED BEFORE CALLING THIS FUNCTION!!

	if (mp_d_colorHistogramImage == 0)
	{
		hipMalloc(&mp_d_colorHistogramImage, width*height * 4);
	}

	if (maxBinCount == 0) maxBinCount = m_max_histogramBinValue;

	BuildHistogramImage_512(mp_d_colorHistogramImage, mp_d_histogram, 512, width, height, maxBinCount);

	hipMemcpy(histImage, mp_d_colorHistogramImage, width * height * 4, hipMemcpyDeviceToHost);
}


void CudaImage::CalculateMaskApertureSums(uint32_t* sums)
{
	if (mp_d_maskApertureSums != 0)	hipFree(mp_d_maskApertureSums);
	uint32_t numApertures = m_maskRows * m_maskCols;
	hipMalloc(&mp_d_maskApertureSums, numApertures * sizeof(uint32_t));
	hipMemset(mp_d_maskApertureSums, 0, numApertures * sizeof(uint32_t));

	CalcApertureSums(mp_d_maskApertureSums, mp_d_grayImage, mp_d_maskImage, m_imageW, m_imageH);

	hipMemcpy(sums, mp_d_maskApertureSums, numApertures * sizeof(uint32_t), hipMemcpyDeviceToHost);
}



void CudaImage::SetFlatFieldCorrectionArrays(int type, float* Gc, float* Dc, int numElements)
{
	// 1 = Fluor
	// 2 = Lumi

	if (type < 1 || type > 2) type = 1;

	m_h_FFC_numElements = (uint32_t)numElements;

	switch (type)
	{
	case 1:
		if (mp_d_FFC_Fluor_Gc != 0)
		{
			hipError_t err = hipFree(mp_d_FFC_Fluor_Gc);
		}
		if (mp_d_FFC_Fluor_Dc != 0)
		{
			hipError_t err = hipFree(mp_d_FFC_Fluor_Dc);
		}

		hipMalloc(&mp_d_FFC_Fluor_Gc, numElements * sizeof(float));
		hipMalloc(&mp_d_FFC_Fluor_Dc, numElements * sizeof(float));

		hipMemcpy(mp_d_FFC_Fluor_Gc, Gc, numElements * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(mp_d_FFC_Fluor_Dc, Dc, numElements * sizeof(float), hipMemcpyHostToDevice);
		break;
	case 2:
		if (mp_d_FFC_Lumi_Gc != 0)
		{
			hipError_t err = hipFree(mp_d_FFC_Fluor_Gc);
		}
		if (mp_d_FFC_Lumi_Dc != 0)
		{
			hipError_t err = hipFree(mp_d_FFC_Fluor_Dc);
		}

		hipMalloc(&mp_d_FFC_Lumi_Gc, numElements * sizeof(float));
		hipMalloc(&mp_d_FFC_Lumi_Dc, numElements * sizeof(float));

		hipMemcpy(mp_d_FFC_Lumi_Gc, Gc, numElements * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(mp_d_FFC_Lumi_Dc, Dc, numElements * sizeof(float), hipMemcpyHostToDevice);
		break;
	}
}



void CudaImage::FlattenImage(int type)
{
	if (mp_d_grayImage == 0) return; // no image to flatten (a call to SetFullGrayscaleImage or SetRoiGrayscaleImage has not been made)

									 // make sure that the flat field corrector is initialized, if not initialize it so that it has no effect on images
	if (m_h_FFC_numElements != (m_imageW*m_imageH))
	{
		m_h_FFC_numElements = m_imageW*m_imageH;
		float* gc = (float*)malloc(m_imageW*m_imageH * sizeof(float));
		float* dc = (float*)malloc(m_imageW*m_imageH * sizeof(float));
		for (int i = 0; i < m_h_FFC_numElements; i++)
		{
			gc[i] = 1.0;
			dc[i] = 0.0;
		}
		SetFlatFieldCorrectionArrays(1, gc, dc, m_h_FFC_numElements);
		SetFlatFieldCorrectionArrays(2, gc, dc, m_h_FFC_numElements);
	}

	switch (type)
	{
	case 0: // no flattening
		break;
	case 1: // Fluor flattening
		FlattenImage(mp_d_grayImage, mp_d_FFC_Fluor_Gc, mp_d_FFC_Fluor_Dc, m_imageW, m_imageH);
		break;
	case 2: // Lumi flattening
		FlattenImage(mp_d_grayImage, mp_d_FFC_Lumi_Gc, mp_d_FFC_Lumi_Dc, m_imageW, m_imageH);
		break;
	}
}


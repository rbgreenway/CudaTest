#include "hip/hip_runtime.h"
#include "CudaImage.h"

#define LOG_CUDA

// Constructor
CudaImage::CudaImage()
{

}

CudaImage::~CudaImage()
{

}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Cuda Kernels

__global__ void Compute_Histogram_512_Cuda(uint32_t* hist, const uint16_t* data, uint16_t width, uint16_t height, uint8_t maxValueBitWidth)
{
	// NOTE: # of bins of histogram must match block size (number of threads in block), and in this case must be 512.
	//		 i.e. the number of threads per block must be the same as the number of bins.

	// maxValueBitWidth = the number of bits needed to represent the max value in the data array.  For example, if the data
	//					  array is built from a 10-bit A-to-D converter, then maxValueBitWidth = 10 since no value will be greather 
	//					  than 2^10.  The minimum value for maxValueBitWidth is driven by the number of bins.  For 256 bins (2^8), 
	//					  the min value is 8.  If bins were 1024 (i.e. 2^10), then the min value for maxValueBitWidth would be 10.

	if (maxValueBitWidth < 8) maxValueBitWidth = 8; // make sure we aren't below the min as described above

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int nThread = threadIdx.y * blockDim.x + threadIdx.x; // index of thread within block
	int nPixel = y * width + x; // index of pixel within image

	if (x >= width) return;
	if (y >= height) return;

	// if image pixel value == 0, don't add it to the histogram.  Pixel that are 0 are pixels that are outside of the mask 
	// and thus should not be part of the histogram
	if (data[nPixel] == 0) return;

	//Create shared buffer size of threads per block and clear it 
	//Size of array equals numBins 
	__shared__ uint32_t tmpHist[512];
	tmpHist[nThread] = 0;
	__syncthreads();


	//based on the value of this pixel, find the correct bin of the local histogram to increment, and then increment it
	uint8_t shift = maxValueBitWidth - 9;
	int binNumber = data[nPixel] >> shift;

	if (binNumber>511)
	{
		binNumber = 511;
	}

	//float f1 = ((float)(data[nPixel]))/1023.0 * 255;
	//uint8_t binNumber = (uint8_t)f1;


	atomicAdd(&(tmpHist[binNumber]), 1);
	__syncthreads();  // wait for all threads in this block to finish so that the local histogram is finished

					  // Update global memory (global histogram)	
	atomicAdd(&(hist[nThread]), tmpHist[nThread]);

}

__global__ void compute_histogram_256_Cuda(uint32_t* hist, const uint16_t* data, uint16_t width, uint16_t height, uint8_t maxValueBitWidth)
{
	// NOTE: # of bins of histogram must match block size (number of threads in block), and in this case must be 256.
	//		 i.e. the number of threads per block must be the same as the number of bins.

	// maxValueBitWidth = the number of bits needed to represent the max value in the data array.  For example, if the data
	//					  array is built from a 10-bit A-to-D converter, then maxValueBitWidth = 10 since no value will be greather 
	//					  than 2^10.  The minimum value for maxValueBitWidth is driven by the number of bins.  For 256 bins (2^8), 
	//					  the min value is 8.  If bins were 1024 (i.e. 2^10), then the min value for maxValueBitWidth would be 10.

	if (maxValueBitWidth < 8) maxValueBitWidth = 8; // make sure we aren't below the min as described above

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int nThread = threadIdx.y * blockDim.x + threadIdx.x; // index of thread within block
	int nPixel = y * width + x; // index of pixel within image

	if (x >= width) return;
	if (y >= height) return;

	//Create shared buffer size of threads per block and clear it 
	//Size of array equals numBins 
	__shared__ uint32_t tmpHist[256];
	tmpHist[nThread] = 0;
	__syncthreads();


	//based on the value of this pixel, find the correct bin of the local histogram to increment, and then increment it
	uint8_t shift = maxValueBitWidth - 8;
	int binNumber = data[nPixel] >> shift;

	if (binNumber>255)
	{
		binNumber = 255;
	}

	//float f1 = ((float)(data[nPixel]))/1023.0 * 255;
	//uint8_t binNumber = (uint8_t)f1;


	atomicAdd(&(tmpHist[binNumber]), 1);
	__syncthreads();  // wait for all threads in this block to finish so that the local histogram is finished

					  // Update global memory (global histogram)	
	atomicAdd(&(hist[nThread]), tmpHist[nThread]);

}

__global__ void MaskImage_Cuda(uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	// this function zeroes out all pixels in image that are not in the mask

	// image - a greyscale image with each pixel being a uint16_t
	// mask - a image where pixels with value>0 will be passed through, and pixels with value==0 will be masked out (set to zero).
	//		  The mask is created where pixels with a value of 1, belong in mask aperture 1.  Pixels with value of 2, belong in 
	//		  mask aperture 2...and so on.  
	// width,height - dimensions of image in pixels

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in array
	uint32_t n = (y * width) + x;

	// apply mask to image
	if (mask[n] == 0)
	{
		// this pixel is not within a mask aperture, so zero it out
		image[n] = 0;
	}
}

__global__ void FlattenImage_Cuda(uint16_t* image, float* Gc, float* Dc, uint16_t width, uint16_t height)
{
	// this function flattens the image using

	// image - a greyscale image with each pixel being a uint16_t	
	// width,height - dimensions of image in pixels
	// Equation:
	//              flattenedImage[n] = (inputImage[n] - Dc[n]) * Gc[n]
	//	
	//
	//  C = corrected image  (Cij = the pixel at column i and row j)	
	//  D = dark image (this is an image taken with no lighting.  it bascially gives the dark current noise)
	//  G = gain
	//  Dc = D corrected to binning size
	//  Gc = G corrected to binning size


	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in array
	uint32_t n = (y * width) + x;

	// adjust pixel to flatten image
	float fval = ((float)image[n] - Dc[n]) * Gc[n];
	if (fval < 0.0f) fval = 0.0f;
	if (fval > 65535.0f) fval = 65535.0f;

	image[n] = (uint16_t)fval;
}


__global__ void ConvertGrayscaleToColor_Cuda(uint8_t* color, uint16_t* gray, uint8_t* redMap, uint8_t* greenMap, uint8_t* blueMap,
	uint16_t width, uint16_t height, uint16_t maxGrayValue, uint16_t scaleLower, uint16_t scaleUpper)
{
	// this function converts a grayscale image to a color image using the provided color map

	// color - destination color image (format is ARGB)
	// gray -  source grayscale image
	// redMap, greenMap, blueMap - arrays (maps) that provide color components for each possible grayscale value. For example,
	//							   if a pixel in the gray image has a value = 100, then the corresponding pixel in the color image
	//							   would have its RGB component values set to redMap[100], greenMap[100], and blueMap[100], respectively.
	// width, height - image dimensions
	// maxGrayValue - the maximum possible grayscale value, i.e. length of color map (length of redMap, greenMap, and blueMap)

	// scaleLower, scaleUpper - these values are used to scale the grayscale value of a pixel before it is converted to color.
	//
	//                         scaleUpper
	//						   ________________
	//	maxGrayValue|         /
	//				|        /
	//				|       /
	//				|      /
	//			0	|_____/____________________ 
	//                   scaleLower
	//
	//  Here's the math:
	//		if (pixelValue < scaleLower) set pixelValue = 0
	//      else if (pixelValue < scaleUpper) set pixelValue = maxGrayValue
	//      else 

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in gray array
	uint32_t nG = (y * width) + x;

	// calculate pixel position in color array
	uint32_t nC = (y * width * 4) + (x * 4);

	// make sure grayscale value is not outside of color maps
	if (gray[nG] > maxGrayValue) gray[nG] = maxGrayValue;

	// scale the value
	uint16_t val = gray[nG];
	if (val < scaleLower) val = 0;
	else if (val >= scaleUpper) val = maxGrayValue;
	else
	{
		float fval = (float)maxGrayValue / (float)(scaleUpper - scaleLower) * (float)(val - scaleLower);
		val = (uint16_t)fval;
	}

	// set pixel component values for color image
	color[nC + 0] = blueMap[val];	// blue
	color[nC + 1] = greenMap[val];	// green
	color[nC + 2] = redMap[val];	// red
	color[nC + 3] = 255;			// alpha

}

__global__ void CopyCudaArrayToD3D9Memory_Cuda(uint8_t *dest, uint8_t *source, uint16_t pitch, uint16_t width, uint16_t height)
{
	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calc position of pixel in cuda array (remember that pitch may not equal width)
	//uint32_t nD = ((height - 1 - y)*pitch) + (x * 4);
	uint32_t nD = (y*pitch) + (x * 4);
	uint32_t nS = (y*width * 4) + (x * 4);

	// copy data
	dest[nD] = source[nS];
	dest[nD + 1] = source[nS + 1];
	dest[nD + 2] = source[nS + 2];
	dest[nD + 3] = source[nS + 3];
}

__global__ void BuildHistogramImage_Cuda(uint8_t* histImage, uint32_t* hist, uint16_t numBins, uint16_t width, uint16_t height, uint32_t maxBinCount)
{
	// this function builds the image for a histogram given by the variable hist.  
	//
	// histImage - the output histogram image.  This is a color image (ARGB, 8 bits per component)
	// hist - is an array which contains the data for the histogram
	// numBins - is the number of bins in the histogram
	// width, height - dimensions of the histImage in pixels
	// maxBinCount - the maximum value that can appear in each bin of the histogram

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate the array index into the histogram image
	uint32_t n = (y * width * 4) + (x * 4);  // ARGB image

											 // calculate the width of each bin in pixels
	uint16_t binWidth = width / numBins;

	// calculate the bin that this pixel belongs in
	uint16_t binNumber = x / binWidth;
	if (binNumber>numBins) binNumber = numBins;

	// calculate height of the bar for his bin
	uint32_t value = hist[binNumber];  // get the height of the bar for this bin
	uint32_t barHeight = (uint32_t)((float)value * (float)height / (float)maxBinCount);  // calculate the bar height in pixels
	if (barHeight > height) barHeight = height; // make sure the bar height in pixels is not greater than the histogram image height

												// determine if this pixel is in the bar or above it (i.e. determine color of pixel)
	if (y < (height - barHeight)) // pixel is above bar (thus pixel is background color...likely white)
	{
		histImage[n + 0] = 220;	// blue
		histImage[n + 1] = 220;	// green
		histImage[n + 2] = 220;	// red
		histImage[n + 3] = 255;	// alpha
	}
	else  // pixel is part of bar, so make it the color of the bar (likely black)
	{
		histImage[n + 0] = 0;	// blue
		histImage[n + 1] = 0;	// green
		histImage[n + 2] = 0;	// red
		histImage[n + 3] = 255;	// alpha
	}
}

__global__ void CalcApertureSums_Cuda(uint32_t* sumArray, uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	// This function calculate the sum of pixels for each aperture of a mask.  It expects that the mask is formated as follows:
	//		mask pixels with a value of 0 belong to no apertures, thus they will not be part of any sum
	//      mask pixels with a value of 1 belong in aperture 1, which is added to the value in sumArray[0]
	//      mask pixels with a value of 2 belong in aperture 2, which is added to the value in sumArray[1]
	//		and so on...

	// sumArray - output array of the sum of pixel values for each aperature.  For example, for a mask with 24x16 (384) apertures, there
	//			  will be 384 values in sumArray
	// image - input grayscale image from which sums are calculated
	// mask  - input mask that is formatted as described in the description above for this function
	// width, height - dimensions of the image and mask in pixels

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in image and mask
	uint32_t n = (y * width) + x;

	// get aperture number from mask
	if (mask[n] > 0) // is this pixel inside of any of the apertures of the mask?
	{ // yes
		atomicAdd(&sumArray[mask[n] - 1], image[n]);
	}

	__syncthreads();  // wait for all threads in this block to finish so that the local histogram is finished

}

__global__ void FlatField_Cuda(uint16_t* image, uint16_t* dark, uint16_t* gain, uint16_t width, uint16_t height)
{
	// this function flat field corrects the given grayscale image. It uses the following function:
	//
	//		C[i,j] = ((R[i,j] - D[i,j]) * m) / (F[i,j] - D[i,j]) = (R[i,j] - D[i,j]) * G[i,j]
	//
	//			where G[i,j] = m / (F[i,j] - D[i,j])
	//
	//				  m = average of F-D
	//
	//		i,j = row,column of pixel in image
	//		C = corrected image
	//		R = raw image
	//		F = flat field reference image (evenly illuminated image, meant to show unevenness of illumination)
	//		D = dark field reference image (image taken with no illumination, meant to show distribution of background)
	//		G = gain

	//	parameters passed into function:
	//	image - grayscale image to be corrected.  This is both the input and output image (the input image is over written)
	//  dark  - this is the dark field image (must be same dimensions as image), probably stored in database
	//  gain  - this is the gain array (must be same dimensions as image), that is calculated elsewhere
	//  width, height - dimensions of image (and dark) in pixels

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in image and dark arrays
	uint32_t n = (y * width) + x;

	image[n] = (image[n] - dark[n]) * gain[n];
}

__global__ void CopyRoiToFullImage_Cuda(uint16_t* full, uint16_t* roi, uint16_t fullW, uint16_t fullH,
	uint16_t  roiX, uint16_t roiY, uint16_t roiW, uint16_t roiH)
{
	// This function is used to copy a ROI image from the camera into a memory space that holds a full frame.
	// It is used when the camera is set up to capture only a part of the CCD (an Region of Interest - ROI), and 
	// since all of the algorithms, kernels, display routines, etc. are set up to handle full frames, this
	// function simply copies the ROI into a full frame.  Pixels outside the ROI are set to zero.

	// calc x,y position of pixel to operate on in the full frame
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside full frame image
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside full frame image

														// make sure we don't try to operate outside the full image
	if (x >= fullW) return;
	if (y >= fullH) return;

	// calculate pixel position in arrays
	uint32_t fullN = (y * fullW) + x;  // index into full frame

									   // calculate x,y position in ROI
	int32_t xr = x - roiX;
	int32_t yr = y - roiY;

	// are we inside ROI?

	if (x >= roiX && x < (roiX + roiW) && y >= roiY && y < (roiY + roiH))
	{
		uint32_t roiN = (yr * roiW) + xr; // index into roi frame

										  // inside ROI
		full[fullN] = roi[roiN];
	}
	else
	{
		// outside ROI
		full[fullN] = 0;
	}
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel Calling Functions

void CudaImage::ConvertGrayscaleToColor(uint8_t* color, uint16_t* gray, uint8_t* redMap, uint8_t* greenMap, uint8_t* blueMap,
	uint16_t width, uint16_t height, uint16_t maxGrayValue, uint16_t scaleLower, uint16_t scaleUpper)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	ConvertGrayscaleToColor_Cuda << <grid, block >> >(color, gray, redMap, greenMap, blueMap, width, height, maxGrayValue, scaleLower, scaleUpper);

}

void CudaImage::CopyRoiToFullImage(uint16_t* full, uint16_t* roi, uint16_t fullW, uint16_t fullH,
	uint16_t  roiX, uint16_t roiY, uint16_t roiW, uint16_t roiH)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = fullW / block.x;
	grid.y = fullH / block.y;
	grid.z = 1;
	CopyRoiToFullImage_Cuda << <grid, block >> >(full, roi, fullW, fullH, roiX, roiY, roiW, roiH);
}

void CudaImage::MaskImage(uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;
	MaskImage_Cuda << <grid, block >> >(image, mask, width, height);
}

void CudaImage::FlattenImage(uint16_t* image, float* Gc, float* Dc, uint16_t width, uint16_t height)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;
	FlattenImage_Cuda << <grid, block >> >(image, Gc, Dc, width, height);
}

void CudaImage::CopyCudaArrayToD3D9Memory(uint8_t* pDest, uint8_t* pSource, uint16_t pitch, uint16_t width, uint16_t height)
{
	hipError_t res = hipDeviceSynchronize();

	dim3 threadsPerBlock(32, 32);  // 32x16 = 512 threads per block	
	dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
	CopyCudaArrayToD3D9Memory_Cuda << <numBlocks, threadsPerBlock >> >(pDest, pSource, pitch, width, height);
}

void CudaImage::ComputeHistogram_512(uint32_t* hist, const uint16_t* data, uint16_t width, uint16_t height, uint8_t maxValueBitWidth)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1; // block size must be 512 = 32 * 16
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	Compute_Histogram_512_Cuda << <grid, block >> >(hist, data, width, height, maxValueBitWidth);
}

void CudaImage::BuildHistogramImage_512(uint8_t* histImage, uint32_t* hist, uint16_t numBins, uint16_t width, uint16_t height, uint32_t maxBinCount)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	BuildHistogramImage_Cuda << <grid, block >> >(histImage, hist, numBins, width, height, maxBinCount);
}

void CudaImage::CalcApertureSums(uint32_t* sumArray, uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	CalcApertureSums_Cuda << <grid, block >> >(sumArray, image, mask, width, height);
}





uint16_t* CudaImage::SetFullGrayscaleImage(uint16_t* grayImage, uint16_t imageWidth, uint16_t imageHeight)
{
	if (imageWidth != m_imageW || imageHeight != m_imageH)
	{
		if (mp_d_grayImage != 0) hipFree(mp_d_grayImage);
		if (mp_d_colorImage != 0) hipFree(mp_d_colorImage);

		m_imageW = imageWidth;
		m_imageH = imageHeight;
		hipError_t res = hipMalloc(&mp_d_grayImage, m_imageW*m_imageH * sizeof(uint16_t));
		res = hipMalloc(&mp_d_colorImage, m_imageW*m_imageH * 4);
	}

	hipError_t err = hipMemcpy(mp_d_grayImage, grayImage, m_imageW*m_imageH * sizeof(uint16_t), hipMemcpyHostToDevice);

	return mp_d_grayImage;
}



uint16_t* CudaImage::SetRoiGrayscaleImage(uint16_t* roiImage, uint16_t imageWidth, uint16_t imageHeight, uint16_t roiWidth, 
							   uint16_t roiHeight, uint16_t roiX, uint16_t roiY)
{
	if (imageWidth != m_imageW || imageHeight != m_imageH)
	{
		if (mp_d_grayImage != 0) hipFree(mp_d_grayImage);
		if (mp_d_colorImage != 0) hipFree(mp_d_colorImage);

		m_imageW = imageWidth;
		m_imageH = imageHeight;
		hipError_t res = hipMalloc(&mp_d_grayImage, m_imageW*m_imageH * sizeof(uint16_t));
		res = hipMalloc(&mp_d_colorImage, m_imageW*m_imageH * 4);
	}

	if (roiWidth != m_roiW || roiHeight != m_roiH || roiX != m_roiX || roiY != m_roiY)
	{
		if (mp_d_roiImage != 0) hipFree(mp_d_roiImage);

		m_roiW = roiWidth;
		m_roiH = roiHeight;
		m_roiX = roiX;
		m_roiY = roiY;
		hipMalloc(&mp_d_roiImage, m_roiW*m_roiH * sizeof(uint16_t));
	}

	hipError_t errNo = hipMemcpy(mp_d_roiImage, roiImage, m_roiW*m_roiH * sizeof(uint16_t), hipMemcpyHostToDevice);

	CopyRoiToFullImage(mp_d_grayImage, mp_d_roiImage, m_imageW, m_imageH, m_roiX, m_roiY, m_roiW, m_roiH);

	return mp_d_grayImage;
}



uint16_t* CudaImage::SetMaskImage(uint16_t* maskImage, uint16_t maskWidth, uint16_t maskHeight, uint16_t maskRows, uint16_t maskCols)
{
	if (m_maskW != maskWidth || m_maskH != maskHeight)
	{
		if (mp_d_maskImage != 0) hipFree(mp_d_maskImage);

		m_maskW = maskWidth;
		m_maskH = maskHeight;
		m_maskRows = maskRows;
		m_maskCols = maskCols;
		hipMalloc(&mp_d_maskImage, m_maskW*m_maskH * sizeof(uint16_t));
	}


	// copy mask image to GPU
	hipMemcpy(mp_d_maskImage, maskImage, m_maskW*m_maskH * sizeof(uint16_t), hipMemcpyHostToDevice);

	m_maskSet = true;

	return mp_d_maskImage;
}



void CudaImage::SetColorMap(uint8_t* redMap, uint8_t* greenMap, uint8_t* blueMap, uint16_t maxPixelValue)
{
	hipError_t res;
	if (mp_d_redMap != 0) hipFree(mp_d_redMap);
	if (mp_d_greenMap != 0) hipFree(mp_d_greenMap);
	if (mp_d_blueMap != 0) hipFree(mp_d_blueMap);

	res = hipMalloc(&mp_d_redMap, maxPixelValue + 1);
	res = hipMalloc(&mp_d_greenMap, maxPixelValue + 1);
	res = hipMalloc(&mp_d_blueMap, maxPixelValue + 1);

	res = hipMemcpy(mp_d_redMap, redMap, maxPixelValue + 1, hipMemcpyHostToDevice);
	res = hipMemcpy(mp_d_greenMap, greenMap, maxPixelValue + 1, hipMemcpyHostToDevice);
	res = hipMemcpy(mp_d_blueMap, blueMap, maxPixelValue + 1, hipMemcpyHostToDevice);

	m_maxPixelValue = maxPixelValue;

	m_colorMapSet = true;
}


uint8_t* CudaImage::ConvertGrayscaleToColor(uint16_t scaleLower, uint16_t scaleUpper)
{
	if (m_colorMapSet && mp_d_grayImage != 0)
	{
		if (mp_d_colorImage == 0) hipMalloc(&mp_d_colorImage, m_imageW*m_imageH * 4);

		ConvertGrayscaleToColor(mp_d_colorImage, mp_d_grayImage, mp_d_redMap, mp_d_greenMap, mp_d_blueMap, m_imageW, m_imageH, m_maxPixelValue, scaleLower, scaleUpper);
	}

	return mp_d_colorImage;
}


void CudaImage::ApplyMaskToImage()
{
	if (m_maskSet)
	{
		MaskImage(mp_d_grayImage, mp_d_maskImage, m_imageW, m_imageH);
	}
}


uint8_t* CudaImage::PipelineFullImage(uint16_t* grayImage, uint16_t imageWidth, uint16_t imageHeight, bool applyMask)
{
	SetFullGrayscaleImage(grayImage, imageWidth, imageHeight);
	if (applyMask) ApplyMaskToImage();
	ConvertGrayscaleToColor(0, m_maxPixelValue);

	return mp_d_colorImage;
}


uint8_t* CudaImage::PipelineRoiImage(uint16_t* roiImage, uint16_t imageWidth, uint16_t imageHeight, 
	uint16_t roiWidth, uint16_t roiHeight, uint16_t roiX, uint16_t roiY, bool applyMask)
{
	SetRoiGrayscaleImage(roiImage, imageWidth, imageHeight, roiWidth, roiHeight, roiX, roiY);
	if (applyMask) ApplyMaskToImage();
	ConvertGrayscaleToColor(0, m_maxPixelValue);

	return mp_d_colorImage;
}


void CudaImage::Init()
{
	mp_d_grayImage = 0;
	mp_d_colorImage = 0;
	mp_d_maskImage = 0;
	mp_d_roiImage = 0;
	mp_d_redMap = 0;
	mp_d_greenMap = 0;
	mp_d_blueMap = 0;
	m_colorMapSet = false;
	m_maskSet = false;
	m_imageW = 0;
	m_imageH = 0;
	m_roiW = 0;
	m_roiH = 0;
	m_roiX = 0;
	m_roiY = 0;
	m_maskW = 0;
	m_maskH = 0;
	m_maskRows = 0;
	m_maskCols = 0;
	m_maxPixelValue = 65535;
	mp_d_histogram = 0;
	mp_d_colorHistogramImage = 0;

	mp_d_FFC_Fluor_Gc = 0;
	mp_d_FFC_Fluor_Dc = 0;
	mp_d_FFC_Lumi_Gc = 0;
	mp_d_FFC_Lumi_Dc = 0;
	m_h_FFC_numElements = 0;


	// not sure why I have to do this, bu
	hipMalloc(&mp_d_grayImage, 10);
	hipMalloc(&mp_d_colorImage, 10);
}


void CudaImage::Shutdown()
{
	if (mp_d_grayImage != 0) {
		hipError_t err = hipFree(mp_d_grayImage);
		mp_d_grayImage = 0;
	}
	if (mp_d_colorImage != 0) {
		hipFree(mp_d_colorImage);
		mp_d_colorImage = 0;
	}
	if (mp_d_maskImage != 0) {
		hipFree(mp_d_maskImage);
		mp_d_maskImage = 0;
	}
	if (mp_d_roiImage != 0) {
		hipFree(mp_d_roiImage);
		mp_d_roiImage = 0;
	}
	if (mp_d_redMap != 0) {
		hipFree(mp_d_redMap);
		mp_d_redMap = 0;
	}
	if (mp_d_greenMap != 0) {
		hipFree(mp_d_greenMap);
		mp_d_greenMap = 0;
	}
	if (mp_d_blueMap != 0) {
		hipFree(mp_d_blueMap);
		mp_d_blueMap = 0;
	}
	if (mp_d_histogram != 0) {
		hipFree(mp_d_histogram);
		mp_d_histogram = 0;
	}
	if (mp_d_colorHistogramImage != 0) {
		hipFree(mp_d_colorHistogramImage);
		mp_d_colorHistogramImage = 0;
	}
	if (mp_d_maskApertureSums != 0) {
		hipFree(mp_d_maskApertureSums);
		mp_d_maskApertureSums = 0;
	}
	if (mp_d_FFC_Fluor_Gc != 0) {
		hipFree(mp_d_FFC_Fluor_Gc);
		mp_d_FFC_Fluor_Gc = 0;
	}
	if (mp_d_FFC_Fluor_Dc != 0) {
		hipFree(mp_d_FFC_Fluor_Dc);
		mp_d_FFC_Fluor_Dc = 0;
	}
	if (mp_d_FFC_Lumi_Gc != 0) {
		hipFree(mp_d_FFC_Lumi_Gc);
		mp_d_FFC_Lumi_Gc = 0;
	}
	if (mp_d_FFC_Lumi_Dc != 0) {
		hipFree(mp_d_FFC_Lumi_Dc);
		mp_d_FFC_Lumi_Dc = 0;
	}
}


void CudaImage::GetHistogram_512Buckets(uint32_t* destHist, uint8_t maxValueBitWidth)
{
	if (mp_d_histogram == 0)
	{
		hipMalloc(&mp_d_histogram, 512 * sizeof(uint32_t));
	}

	hipMemset(mp_d_histogram, 0, 512 * sizeof(uint32_t));

	ComputeHistogram_512(mp_d_histogram, mp_d_grayImage, m_imageW, m_imageH, maxValueBitWidth);

	//hipMemset(mp_d_histogram, 0, sizeof(uint32_t));  // zero the first bin, since that is the pixels that were masked out

	hipMemcpy(destHist, mp_d_histogram, 512 * sizeof(uint32_t), hipMemcpyDeviceToHost);

	m_max_histogramBinValue = 0;

	for (int i = 1; i < 512; i++)
	{
		if (destHist[i] > m_max_histogramBinValue) m_max_histogramBinValue = destHist[i];
	}

}


void CudaImage::GetHistogramImage_512Buckets(uint8_t* histImage, uint16_t width, uint16_t height, uint32_t maxBinCount)
{
	// NOTE:  GetHistogram_512Buckets MUST BE CALLED BEFORE CALLING THIS FUNCTION!!

	if (mp_d_colorHistogramImage == 0)
	{
		hipMalloc(&mp_d_colorHistogramImage, width*height * 4);
	}

	if (maxBinCount == 0) maxBinCount = m_max_histogramBinValue;

	BuildHistogramImage_512(mp_d_colorHistogramImage, mp_d_histogram, 512, width, height, maxBinCount);

	hipMemcpy(histImage, mp_d_colorHistogramImage, width * height * 4, hipMemcpyDeviceToHost);
}


void CudaImage::CalculateMaskApertureSums(uint32_t* sums)
{
	if (mp_d_maskApertureSums != 0)	hipFree(mp_d_maskApertureSums);
	uint32_t numApertures = m_maskRows * m_maskCols;
	hipMalloc(&mp_d_maskApertureSums, numApertures * sizeof(uint32_t));
	hipMemset(mp_d_maskApertureSums, 0, numApertures * sizeof(uint32_t));

	CalcApertureSums(mp_d_maskApertureSums, mp_d_grayImage, mp_d_maskImage, m_imageW, m_imageH);

	hipMemcpy(sums, mp_d_maskApertureSums, numApertures * sizeof(uint32_t), hipMemcpyDeviceToHost);
}



void CudaImage::SetFlatFieldCorrectionArrays(int type, float* Gc, float* Dc, int numElements)
{
	// 1 = Fluor
	// 2 = Lumi

	if (type < 1 || type > 2) type = 1;

	m_h_FFC_numElements = (uint32_t)numElements;

	switch (type)
	{
	case 1:
		if (mp_d_FFC_Fluor_Gc != 0)
		{
			hipError_t err = hipFree(mp_d_FFC_Fluor_Gc);
		}
		if (mp_d_FFC_Fluor_Dc != 0)
		{
			hipError_t err = hipFree(mp_d_FFC_Fluor_Dc);
		}

		hipMalloc(&mp_d_FFC_Fluor_Gc, numElements * sizeof(float));
		hipMalloc(&mp_d_FFC_Fluor_Dc, numElements * sizeof(float));

		hipMemcpy(mp_d_FFC_Fluor_Gc, Gc, numElements * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(mp_d_FFC_Fluor_Dc, Dc, numElements * sizeof(float), hipMemcpyHostToDevice);
		break;
	case 2:
		if (mp_d_FFC_Lumi_Gc != 0)
		{
			hipError_t err = hipFree(mp_d_FFC_Fluor_Gc);
		}
		if (mp_d_FFC_Lumi_Dc != 0)
		{
			hipError_t err = hipFree(mp_d_FFC_Fluor_Dc);
		}

		hipMalloc(&mp_d_FFC_Lumi_Gc, numElements * sizeof(float));
		hipMalloc(&mp_d_FFC_Lumi_Dc, numElements * sizeof(float));

		hipMemcpy(mp_d_FFC_Lumi_Gc, Gc, numElements * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(mp_d_FFC_Lumi_Dc, Dc, numElements * sizeof(float), hipMemcpyHostToDevice);
		break;
	}
}



void CudaImage::FlattenImage(int type)
{
	if (mp_d_grayImage == 0) return; // no image to flatten (a call to SetFullGrayscaleImage or SetRoiGrayscaleImage has not been made)

									 // make sure that the flat field corrector is initialized, if not initialize it so that it has no effect on images
	if (m_h_FFC_numElements != (m_imageW*m_imageH))
	{
		m_h_FFC_numElements = m_imageW*m_imageH;
		float* gc = (float*)malloc(m_imageW*m_imageH * sizeof(float));
		float* dc = (float*)malloc(m_imageW*m_imageH * sizeof(float));
		for (int i = 0; i < m_h_FFC_numElements; i++)
		{
			gc[i] = 1.0;
			dc[i] = 0.0;
		}
		SetFlatFieldCorrectionArrays(1, gc, dc, m_h_FFC_numElements);
		SetFlatFieldCorrectionArrays(2, gc, dc, m_h_FFC_numElements);
	}

	switch (type)
	{
	case 0: // no flattening
		break;
	case 1: // Fluor flattening
		FlattenImage(mp_d_grayImage, mp_d_FFC_Fluor_Gc, mp_d_FFC_Fluor_Dc, m_imageW, m_imageH);
		break;
	case 2: // Lumi flattening
		FlattenImage(mp_d_grayImage, mp_d_FFC_Lumi_Gc, mp_d_FFC_Lumi_Dc, m_imageW, m_imageH);
		break;
	}
}